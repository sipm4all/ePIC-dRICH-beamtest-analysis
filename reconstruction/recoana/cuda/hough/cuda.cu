// cuda.cu
#include <hip/hip_runtime.h>
#include <stdio.h> 

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
	    file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/** 
    Hough space GPU model
    the basic computational unit is a 10x10x10 block in the (x,y,r) space
    the lattice is defined by the cell spacing (dx,dy,dr)

    the full Hough space can be mapped with Nx,Ny,Nr blocks

    for example, if we want to map a Hough space 
    x = [-64, 64]
    y = [-64, 64]
    r = [ 32, 96]
    with a cell spacing of (1,1,1) we need (Nx,Ny,Nr) = (16,16,8) blocks
    Nx = 128 / dx / 8 = 16
    Ny = 128 / dy / 8 = 16
    Nr =  64 / dr / 8 = 8

**/

float *gpu_x = nullptr;
float *gpu_y = nullptr;
float *gpu_h = nullptr;
float *gpu_rh = nullptr;
int *gpu_rhi = nullptr;

float *gpu_xmap = nullptr;
float *gpu_ymap = nullptr;
float *gpu_rmap = nullptr;

const float x_min = -15.5;
const float x_stp = 1.;
const float y_min = -15.5;
const float y_stp = 1.;
const float r_min = 32.;
const float r_stp = 1.;

__global__ void
hough_gpu_init(float *xmap, float *ymap, float *rmap, int Nx, int Ny, int Nr) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int ix = threadIdx.x % 8;
  int iy = (threadIdx.x / 8) % 8;
  int ir = threadIdx.x / 64;
  
  int iX = blockIdx.x % Nx;
  int iY = (blockIdx.x / Nx) % Ny;
  int iR = blockIdx.x / (Nx * Ny);
  
  ix += iX * 8;
  iy += iY * 8;
  ir += iR * 4;
  
  float x = x_min + ix * x_stp;
  float y = y_min + iy * y_stp;
  float r = r_min + ir * r_stp;

  xmap[tid] = x;
  ymap[tid] = y;
  rmap[tid] = r;
  
}

__global__ void
hough_gpu_transform(float *xmap, float *ymap, float *rmap, float *x, float *y, float *h, int n)
{

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  float cx = xmap[tid];
  float cy = ymap[tid];
  float cr = rmap[tid];

  h[tid] = 0.;
  for (int i = 0; i < n; ++i) {
    float dx = cx - x[i];
    float dy = cy - y[i];
    float dr = hypotf(dx, dy) - cr;
    float w = 0.11398351 * expf(-0.040816327 * dr * dr  );
    h[tid] += w;
  }
  
}

__global__ void
find_max_kernel(float *h, float *rh, int *rhi)
{
  __shared__ float shm[256];
  __shared__ int shmi[256];
  
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  shm[tid] = h[gid];
  shmi[tid] = gid;
  __syncthreads();

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      if (shm[tid + stride] > shm[tid]) {
	shm[tid] = shm[tid + stride];
	shmi[tid] = shmi[tid + stride];
      }
    }
    __syncthreads();
  }

  if (tid == 0) {
    rh[bid] = shm[0];
    rhi[bid] = shmi[0];
  }

}

void
hough_init(float *cpu_xmap, float *cpu_ymap, float *cpu_rmap, int Nx, int Ny, int Nr)
{
  int Nh = 256 * Nx * Ny * Nr;
  
  // alloc device memory
  HANDLE_ERROR( hipMalloc((void **)&gpu_x, 1024 * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&gpu_y, 1024 * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&gpu_h, Nh * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&gpu_rh, Nx * Ny * Nr * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&gpu_rhi, Nx * Ny * Nr * sizeof(int)) );

  HANDLE_ERROR( hipMalloc((void **)&gpu_xmap, Nh * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&gpu_ymap, Nh * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&gpu_rmap, Nh * sizeof(float)) );

  // launch kernel
  dim3 block_size(256, 1, 1);
  dim3 grid_size(Nx * Ny * Nr, 1, 1);
  hough_gpu_init<<<grid_size, block_size>>>(gpu_xmap, gpu_ymap, gpu_rmap, Nx, Ny, Nr);

  // copy data from device
  HANDLE_ERROR( hipMemcpy(cpu_xmap, gpu_xmap, Nh * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(cpu_ymap, gpu_ymap, Nh * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(cpu_rmap, gpu_rmap, Nh * sizeof(float), hipMemcpyDeviceToHost) );

}

void
hough_free()
{
  // free device memory
  hipFree(gpu_x);
  hipFree(gpu_y);
  hipFree(gpu_h);
  hipFree(gpu_rh);
  hipFree(gpu_rhi);
  
  hipFree(gpu_xmap);
  hipFree(gpu_ymap);
  hipFree(gpu_rmap);
}

void
hough_transform(float *cpu_x, float *cpu_y, float *cpu_rh, int *cpu_rhi, int cpu_n, int Nx, int Ny, int Nr)
{
  int Nrh = Nx * Ny * Nr;

  // copy data to device
  HANDLE_ERROR( hipMemcpy(gpu_x, cpu_x, cpu_n * sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(gpu_y, cpu_y, cpu_n * sizeof(float), hipMemcpyHostToDevice) );
  
  // launch kernel
  dim3 block_size(256, 1, 1);
  dim3 grid_size(Nx * Ny * Nr, 1, 1);
  hough_gpu_transform<<<grid_size, block_size>>>(gpu_xmap, gpu_ymap, gpu_rmap, gpu_x, gpu_y, gpu_h, cpu_n);
  find_max_kernel<<<grid_size, block_size>>>(gpu_h, gpu_rh, gpu_rhi);
  
  // copy data from device
  HANDLE_ERROR( hipMemcpy(cpu_rh, gpu_rh, Nrh * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(cpu_rhi, gpu_rhi, Nrh * sizeof(int), hipMemcpyDeviceToHost) );
}

